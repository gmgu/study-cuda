
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_kernel() {
  printf("hello kernel!\n");
}

int main() {
  hello_kernel<<<1, 2>>>();
  hipDeviceSynchronize();
}
