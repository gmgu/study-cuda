#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


using namespace std;

#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < N)
        out[tid] = a[tid] + b[tid];
}

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    int num_block = (int)(N / 256);
    if(N % 256 != 0)
        num_block += 1;

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Max grid: %d\n", prop.maxGridSize[0]);
    printf("Max grid: %d\n", prop.maxGridSize[1]);
    printf("Max grid: %d\n", prop.maxGridSize[2]);
    printf("Max Thread: %d %d %d %d\n", prop.maxThreadsPerBlock, prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("#multiprocessors: %d\n", prop.multiProcessorCount);

    // Executing kernel 
    vector_add<<<num_block,256>>>(d_out, d_a, d_b, N);
    
    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Verification
    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);
}
